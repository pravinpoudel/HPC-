
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>

char* readFile(char* url){

    int fileSize, readCount;
    char* buffer;

    FILE *filePointer;
    filePointer = fopen(url, "r");

    if(filePointer == NULL){
        printf("failed to open the file");
    }

    else{
        printf("opened the file successfully");

// use pointer to find the fileSize by going to the end and rewind it to the start of the file
//after work is done

        fseek(filePointer, 0, SEEK_END);
        fileSize = ftell(filePointer);
        rewind(filePointer);

//we wanted that size to allocate a memory for a string of that size, which is job 
//of buffer

//i am adding one more character size to add /0 at the end
    buffer = (char*) malloc(sizeof(char) * (fileSize + 1));

    readCount = fread(buffer, sizeof(char), fileSize, filePointer);

    buffer[fileSize] = '\0';

    if(fileSize  != readCount){
        free(buffer);
        printf("there was some errror reading the file, please try again");
    }

    fclose(filePointer);

    }

    return buffer;

    }


int main(){

    char *fileString = readFile("raw file");



}







