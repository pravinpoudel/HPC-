#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

__global__ void cuda_hello(){
    printf("Hello world from cuda device");
}

int main(){
    cuda_hello<<1,1>>()
    hipDeviceSynchronize();
    return 0;
}